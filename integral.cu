
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>

__device__ float myPower(float* number, int degree) {
  float result = 1.0;
  int fraction = 0;


  if(degree == 0) {
    return result;
  } else if( degree < 0) {
    degree = degree * (-1);
    fraction = 1;
  }

  for(int i = 1; i <= degree; i++) {
    result *= (*number);
  }

  if(fraction == 0) {
    return result;
  } else  {
    return 1/result;
  }
}
/**
 * @brief Caclulate polynomial function from given data.
 * @param x Function input value
 * @param function Polynomial function data.
 */
__device__ float function(float* x, float* coefficients, unsigned int polynomialDegree) {
   unsigned int polynomialItertor = 0;
   float functionResult = 0;
   float tmpCalc;
   for(polynomialItertor = 0; polynomialItertor <= polynomialDegree; polynomialItertor++) {
     tmpCalc = coefficients[polynomialItertor] * myPower(x,polynomialItertor);
     functionResult += tmpCalc;
   }
   return functionResult;
}

__host__ float functionHost(float x, float* coefficients, unsigned int polynomialDegree) {
   unsigned int polynomialItertor = 0;
   float functionResult = 0;
   float tmpCalc;
   for(polynomialItertor = 0; polynomialItertor <= polynomialDegree; polynomialItertor++) {
     tmpCalc = coefficients[polynomialItertor] * pow(x,polynomialItertor);
     functionResult += tmpCalc;
   }
   return functionResult;
}

__global__ void numericalIntegrationArray(float* coefficients, unsigned int polynomialDegree, float* xArray_device, float* yArray_device, int N) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if( i < N) {
    yArray_device[i] = function(&xArray_device[i],coefficients,polynomialDegree);
  }
}

int GPU_Integration(float* coefficients, unsigned int polynomialDegree, float low, float high, float precision, float *result, int nThx) {
   int numberOfPoints = (int) (high-low) / precision;
   int sizeOfArray = sizeof(float)*numberOfPoints;
   float *array, *xArray_device, *yArray_device;
   hipEvent_t start, stop;

   array = (float*) malloc(sizeOfArray);
   for(int i = 0; i < numberOfPoints; i++) {
     array[i] = low+i*precision;
   }
   hipMalloc((void**)&xArray_device, sizeOfArray);
   hipMemcpy(xArray_device, array, sizeOfArray,hipMemcpyHostToDevice);

   hipMalloc((void**)&yArray_device, sizeOfArray);

   float* coefficients_d;
   hipMalloc((void**) &coefficients_d, sizeof(float)*(polynomialDegree+1));
   hipMemcpy(coefficients_d, coefficients, sizeof(float)*(polynomialDegree+1), hipMemcpyHostToDevice);

   int nBLK = (int)(numberOfPoints+nThx-1)/nThx;

   printf("    GPU integral with parameter : \n");
   printf("    Number of blocks: %d\n", nBLK);
   printf("    Number of thread per block: %d\n", nThx);
   printf("    Precision of integral calculation %f\n", precision);

   float time;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   hipEventRecord(start, 0);

   numericalIntegrationArray<<<nBLK,nThx>>>(coefficients_d,polynomialDegree,xArray_device,yArray_device,sizeOfArray);
   hipDeviceSynchronize();

   //for (int n = ceil((float)sizeOfArray/2.); n > 2; n = ceil((float)sizeOfArray/2.)) {
   //}

printf("sizeofArray = %d", sizeOfArray);
   hipEventRecord(stop, 0);
   hipEventSynchronize(stop);
   hipEventElapsedTime(&time, start, stop);
   printf("  GPU time is %f ms\n", time);

   hipMemcpy(array, yArray_device, sizeOfArray, hipMemcpyDeviceToHost);
   hipFree(coefficients_d);
   hipFree(xArray_device);
   hipFree(yArray_device);


   for(int j = 0; j < numberOfPoints ; j++  ) {
     *result += array[j];
   }

   *result *= precision;

   free(array);

   return 0;
}

int main(void) {
  const unsigned int polynomial1Size = 2;
  const unsigned int polynomial2Size = 5;
  float polynomial1[3];

  polynomial1[0] = 1.25;
  polynomial1[1] = 2.5;
  polynomial1[2] = 1.0;

  float polynomial2[6];
  polynomial2[0] = 3.1;
  polynomial2[1] = 2.5;
  polynomial2[2] = 1.3;
  polynomial2[3] = 10.1;
  polynomial2[4] = 54.0;
  polynomial2[5] = 1.25;

  float lowData = 0.0;
  float highData = 4.0;
  float prec1 = 0.001;
  float prec2 = 0.0001;

  float result1 = 0.0;
  float result2 = 0.0;
  float result3 = 0.0;
  float result4 = 0.0;

  float result11 = 0.0;
  float result21 = 0.0;
  float result31 = 0.0;
  float result41 = 0.0;

  int nThx = 128;
  int nThx1 = 256;

  printf("Function 1:");
  GPU_Integration(polynomial1, polynomial1Size, lowData, highData, prec1, &result1, nThx);
  printf("  Result %f\n", result1);

  printf("Function 1:");
  GPU_Integration(polynomial1, polynomial1Size, lowData, highData, prec1, &result2, nThx1);
  printf("  Result %f\n", result2);

  printf("Function 1:");
  GPU_Integration(polynomial1, polynomial1Size, lowData, highData, prec2, &result3, nThx);
  printf("  Result %f\n", result3);

  printf("Function 1:");
  GPU_Integration(polynomial1, polynomial1Size, lowData, highData, prec2, &result4, nThx1);
  printf("  Result %f\n", result4);


  printf("Function 2:");
  GPU_Integration(polynomial2, polynomial2Size, lowData, highData, prec1, &result11, nThx);
  printf("  Result %f\n", result11);

  printf("Function 2:");
  GPU_Integration(polynomial2, polynomial2Size, lowData, highData, prec1, &result21, nThx1);
  printf("  Result %f\n", result21);

  printf("Function 2:");
  GPU_Integration(polynomial2, polynomial2Size, lowData, highData, prec2, &result31, nThx);
  printf("  Result %f\n", result31);

  printf("Function 2:");
  GPU_Integration(polynomial2, polynomial2Size, lowData, highData, prec2, &result41, nThx1);
  printf("  Result %f\n", result41);
}
